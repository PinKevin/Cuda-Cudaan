#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>

// Fungsi CUDA untuk menghitung jumlah bilangan genap dalam rentang tertentu
__global__ void HitungJumlahBagian(int start, int end, int* hasil) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;

    int JumlahBagian = 0;
    for (int i = start + tid; i <= end; i += step) {
        if (i % 2 == 0) {
            JumlahBagian += i;
        }
    }

    atomicAdd(hasil, JumlahBagian);
}

int main() {
    int N;
    srand(static_cast<unsigned int>(time(nullptr)));
    N = rand() % 1000000 + 1;

    int* d_hasil;
    hipMalloc(&d_hasil, sizeof(int));
    hipMemset(d_hasil, 0, sizeof(int));

    // Konfigurasi blok dan grid CUDA
    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;

    // Menghitung waktu awal eksekusi
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Bagi perhitungan ke dalam beberapa bagian
    int numParts = 10; // Misalnya, kita bagi menjadi 300 bagian
    int partSize = N / numParts;

    int totalHasil = 0; // Variabel untuk menghitung total hasil

    for (int i = 0; i < numParts; ++i) {
        int startPart = i * partSize + 1;
        int endPart = (i != numParts - 1) ? (i + 1) * partSize : N;

        // Memanggil kernel CUDA untuk menghitung jumlah bagian
        HitungJumlahBagian << <gridSize, blockSize >> > (startPart, endPart, d_hasil);
        hipDeviceSynchronize(); // Menunggu kernel selesai

        // Mengambil hasil dari perangkat CUDA
        int hasil;
        hipMemcpy(&hasil, d_hasil, sizeof(int), hipMemcpyDeviceToHost);

        std::cout << "Bagian " << i + 1 << ": Jumlah bilangan genap dari " << startPart << " hingga " << endPart << " adalah " << hasil << std::endl;

        totalHasil += hasil; // Menambahkan hasil sementara ke total
    }

    // Menghitung waktu akhir eksekusi
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << "Hasil total penghitungan: " << totalHasil << std::endl;
    std::cout << "Waktu eksekusi total: " << milliseconds / 1000.0 << " detik" << std::endl; // Konversi ke detik

    // Membebaskan memori GPU
    hipFree(d_hasil);

    return 0;
}
